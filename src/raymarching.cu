#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "raymarching.h"
//__global__ void render(maths::Vector3f *framebuffer, int max_x, int max_y, float fov)
//{
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	int j = threadIdx.y + blockIdx.y * blockDim.y;
//	if ((i >= max_x) || (j >= max_y)) return;
//	int pixel_index = j * max_x + i;
//	double dir_x = (j + 0.5f) - max_x / 2.0;
//	double dir_y = -(i + 0.5f) + max_y / 2.0;
//	double dir_z = -max_y / (2.0 * tan(fov/ 2.0));
//	int pixelIndex = j * max_x + i;
//	maths::Vector3f ray_direction = maths::Vector3f(dir_x, dir_y, dir_z).Normalized();
//}
//
//int main()
//{
//	int nx = 1920;
//	int ny = 1080;
//int num_pixels = nx* ny;
////size_t fb_size = 3 * num_pixels * sizeof(float);
////
////// allocate FB
////float* frame_buffer;
////hipError_t cudaStatus;
////hipMallocManaged((void**)&frame_buffer, fb_size);
////
////int tx = 8;
////int ty = 8;
////
////dim3 blocks(nx / tx + 1, ny / ty + 1);
////dim3 threads(tx, ty);
////render <<<blocks, threads >>> (frame_buffer);
//
///// <summary>
///// ////////
///// </summary>
/////
//raytracing::RayMarcher raymarcher;
//
//
//std::vector<maths::Vector3f> frameBuffer(num_pixels);
//
//maths::Vector3f* d_fb;
//
//hipMallocManaged(&d_fb, num_pixels * sizeof(maths::Vector3f));
//
//maths::Sphere sphere1(1.0f, maths::Vector3f(0.0f, 0.0f, -10.0f));
//maths::Sphere sphere2(1.0f, maths::Vector3f(3.0f, 0.0f, -10.0f));
//
//	
//std::vector<maths::Sphere> h_sphere;
//h_sphere.push_back(sphere1);
//h_sphere.push_back(sphere2);
//
//maths::Sphere* d_spheres;
//hipMalloc(&d_spheres, h_sphere.size() * sizeof(maths::Sphere));
//
//hipMemcpy(d_spheres, h_sphere.data(), h_sphere.size() * sizeof(maths::Sphere), hipMemcpyHostToDevice);
//
//
//return 0;
//}
//
//
//



